using namespace std;

class flight
{
	private:
		
		string flightNumber;
		string departureDateAndTime;
		string departureAirport;
		string arrivalAirport;
		char seats[60][6]; //to keep seat details
	
	public:
		
		flight(); //constructor
		
		//member functions
		void set_flightNumber(string str){flightNumber=str;}
		void set_departureDateAndTime(string str){departureDateAndTime=str;}
		void set_departureAirport(string str){departureAirport=str;}
		void set_arrivalAirport(string str){arrivalAirport=str;}
		void set_seats(int row,int column,char value)
		{
			seats[row][column]=value;
		}
		
		//friend functions
		friend int showAvailableFlights(); //option 1
		friend void viewFlight(); //option 2
		friend int seatAvailability(); //option 3
		friend int seatBooking(); //option 4
		friend void exit(); //option 5
};

vector<flight> database; //vector of objects, (to store flight objects) -- dynamically --

flight::flight() //initializing the whole seat array to '/' values
{
	for(int i=0;i<60;i++) 
	{
		for(int j=0;j<6;j++)
		{
			seats[i][j]='/';
		}
	}	
}

void initialize()
{ 
//function which runs in the begining
//opens the text file and store all the data to database 
	 
	flight object; //creating a temporary object 
	int line=0,i=0; //line variable is used to identify attributes one by one and i variable is used to seperate flight by flight
	int  row=0,column=0,count=0; //to handle seat 2D array's data storing process
	char seatClass;	//to store the class of a seat
	
	//file operations
	
	fstream flights; 
	flights.open("flights.txt", ios::in); //opening the file
	flight x;
	if(!flights)
	{
		cout<<"No such file"; //if the file is not available
	} 
	else
	{ 
  		ifstream file("flights.txt");
  		string str;
  		while (getline(file,str)) //read line by line
		{
			if(line==0) //store flight number
			{
				object.set_flightNumber(str); //adding flight number data to the object temporarily
			}
			else if(line==1) //store departure Date And Time
			{
				object.set_departureDateAndTime(str); //adding departure Date And Time data to the object temporarily
			}
			else if(line==2) //store departure Airport
			{
				object.set_departureAirport(str); //adding departure Airport data to the object temporarily
			}
			else if(line==3) //store arrival Airport
			{
				object.set_arrivalAirport(str); //adding arrival Airport data to the object temporarily
			}
			else //to store available seat data into the 2D array
			{
				count=0; //to get word by word
    			istringstream substring(str);  	// Used to split the string
				string word; 
   				do { 	//getting word by word from the large string
        			
        			substring>>word; 
					if(count==0) //to get row number
					{
						istringstream(word)>>row; //converting it to int and assigning to row
				
					} 
        			else if(count==1) //to get the class of the seat
					{
						seatClass=word[0];
					
					} 
					else if(count==2) // to get the seat columns
					{
						char seatColumn;
						for(int j=0;j<word.size();j++)
						{
							seatColumn=word[j]; //getting character by character from the word
		
							column=(int)seatColumn; //converting to integer
							column=column-65; //I store these in the 2D array like this--> 0 column for A, 1 for B ... 5 for E
							
    						object.set_seats(row-1,column,seatClass);
						}
					}
        			count++;
    				}while(substring); 
			}
			
			line++;
			if(str=="")
			{
    			line=0; 
    			database.push_back(object);  //after allocating all the data to object, appending the object to the back of the vector
    			
    			//to clean previous object's records about seats
				for(int j=0;j<60;j++)
				{
					for(int l=0;l<6;l++)
					{
						object.set_seats(j,l,'/');
					}	
				}
				
				i++;
			}
			
  		}
	}
    flights.close(); 

}


int showAvailableFlights()
{
	cout<<"--Available Flights Details--"<<endl<<endl;
	int bigFlag=0,smallFlag=0,economySeatCount=0,businessSeatCount=0;
	
	for(int i=0;i<database.size();i++) //the flight
	{
		smallFlag=0;
		economySeatCount=0;
		businessSeatCount=0;
		
		for(int j=0;j<60;j++) //the row
		{
			for(int l=0;l<6;l++) //the column
			{
				if(database[i].seats[j][l]!='/')
				{
					bigFlag=1; //to find are there any available flight
					smallFlag=1; //to identify available flights one by one
					if(database[i].seats[j][l]=='E')
					{
						economySeatCount++;
					}
					else if(database[i].seats[j][l]=='B')
					{
						businessSeatCount++;
					}
				}
			}
		}
		if(smallFlag)
		{
			cout<<"Flight Number: "<<database[i].flightNumber<<endl;
			cout<<"Flight Departure Date and Time: "<<database[i].departureDateAndTime<<endl;
			cout<<"Departure Airport: "<<database[i].departureAirport<<endl;
			cout<<"Arrival Airport: "<<database[i].arrivalAirport<<endl;
			cout<<"--Available Seats--"<<endl;
			cout<<"In Business Class: "<<businessSeatCount<<" | In Economy Class: "<<economySeatCount<<endl<<endl;
		}
		cout<<endl;
	}
	if(!bigFlag)
	{
		cout<<"there are no available filghts";
		return 0;
	}
	
}

void viewFlight()
{
	string fNum;
	int flag=0;
	cout<<"insert the flight number: ";
	cin>>fNum;
	for(int i=0;i<database.size();i++)
	{
		if(database[i].flightNumber==fNum) //check whether the flight num is available
		{
			flag=1;
			int economySeatCount=0;
			int businessSeatCount=0;
			for(int j=0;j<60;j++) //the row
			{
				for(int l=0;l<6;l++) //the column
				{
					if(database[i].seats[j][l]!='/') //available seats
					{
						printf("Seat Number: %d-%c | Seat Class: %c\n",j+1,l+65,database[i].seats[j][l]);	
						if(database[i].seats[j][l]=='E')
						{
							economySeatCount++;
						}
						else if(database[i].seats[j][l]=='B')
						{
							businessSeatCount++;
						}
					}
				}	
			}
			cout<<"Available Economy Class Seats in the flight:"<<economySeatCount<<endl;
			cout<<"Available Business Class Seats in the flight:"<<businessSeatCount<<endl;	
			cout<<"Departure Airport: "<<database[i].departureAirport<<endl;
			cout<<"Arrival Airport: "<<database[i].arrivalAirport<<endl;
			cout<<"Flight Departure Date and Time: "<<database[i].departureDateAndTime<<endl;
			break;
		}
	}
	if(!flag)
	{
		cout<<"invalid flight number or flight is not present";
	}
	
}

int seatAvailability()
{
	string fNum;
	int seatAmount=0,flag=0,seatCount=0; //seatAmount->seat amount user wants , seatCount->seat amount that particular flight has.
	cout<<"Insert the flight number: ";
	cin>>fNum;
	cout<<"Insert the number of seats required: ";
	cin>>seatAmount;
	
	//to count the availble seats
	for(int i=0;i<database.size();i++)
	{
		if(database[i].flightNumber==fNum)
		{
			flag=1;
			for(int j=0;j<60;j++) //the row
			{
				for(int l=0;l<6;l++) //the column
				{
					if(database[i].seats[j][l]!='/') //available seats
					{
						seatCount++;
					}
				}	
			}
			break;
		}
	}
	//when user inputs invalid flight number
	if(!flag)
	{
		cout<<"Flight name invalid or flight is not present";
		return 0;
	}
	//when there are no enough seats
	if(seatCount<seatAmount)
	{
		cout<<"Not enough seats";
	}
	//if there are enough seats
	else
	{
		cout<<"\n\n---Seat Numbers That Are Available---\n\n";
		for(int i=0;i<database.size();i++)
		{
			if(database[i].flightNumber==fNum)
			{
				for(int j=0;j<60;j++) //the row
				{
					for(int l=0;l<6;l++) //the column
					{
						if(database[i].seats[j][l]!='/') //available seats
						{
							printf("Seat Number: %d-%c\n",j+1,l+65);
						}
					}	
				}
				break;
			}
		}
	}
}

int seatBooking()
{
	string fNum;
	int row,intColumn,bigFlag=0,smallFlag=0;
	char column; 
	cout<<"Insert the flight number: ";
	cin>>fNum;
	cout<<"Insert the seat row: ";
	cin>>row;
	cout<<"Insert the seat column: ";
	cin>>column;
	
	for(int i=0;i<database.size();i++)
	{
		if(database[i].flightNumber==fNum)
		{
			bigFlag=1;	
			if(database[i].seats[row-1][column-65]!='/') //availability
			{
				smallFlag=1;
				database[i].seats[row-1][column-65]='/';
			}
		}	
	}
	//when user inputs invalid flight number
	if(!bigFlag)
	{
		cout<<"Flight name invalid or flight is not present";
		return 0;
	}
	if(!smallFlag)
	{
		cout<<"Seat name is invalid or seat is not available";
		return 0;
	}
	
}

void exit()
{
 	string str1,str2;
 	char fClass;
 	int flag=0;
    // open a file in write mode.
    ofstream toWrite;
    toWrite.open("flights.dat");
    
    // write data into the file.
    
	for(int i=0;i<database.size();i++) //particular flight
	{	
		toWrite<<database[i].flightNumber<<endl;
		toWrite<<database[i].departureDateAndTime<<endl;
		toWrite<<database[i].departureAirport<<endl;
		toWrite<<database[i].arrivalAirport<<endl;	
		
		//write seat data
		for(int j=0;j<60;j++) //the row
		{
			flag=0;
			str1="";
			for(int l=0;l<6;l++) //the column
			{
				if(database[i].seats[j][l]!='/') //only write free seats
				{
					fClass=database[i].seats[j][l]; //assigning class of the flight to a char variable
					flag=1; 
					str2=l+65; //column of the seat
					str1.append(str2); 			
				}
			}
			if(flag)
			{
				//writing seat details
				toWrite<<j+1<<" "<<fClass<<" "<<str1<<endl;
			}
		}
		toWrite<<endl; //blank line that seperates flights
	}
 	toWrite.close();
}


int main()
{
	//Driver Programme
	initialize(); 
	int optionNumber;
	cout<<"~~ Virgin Airline Flight Reservation System ~~"<<endl;
	while(1)
	{
		
		cout<<"\n\n`````` The Main Menu ``````"<<endl;
		cout<<"---------------------------"<<endl;
		cout<<"1.Display Available Flights \n2.View Flight \n3.Seat Availability \n4.Seat Booking \n5.Exit\n";
		cout<<"---------------------------"<<endl;
		cout<<"\nEnter The Option Number: ";	
		cin>>optionNumber;
		cout<<endl;
		
		if(optionNumber==1)
		{
			showAvailableFlights();
		}
		else if(optionNumber==2)
		{
			viewFlight();
		}
		else if(optionNumber==3)
		{
			seatAvailability(); 
		}
		else if(optionNumber==4)
		{
			seatBooking();
		}
		else if(optionNumber==5)
		{
			exit();
			break;
		}
		else
		{
			cout<<"Invalid Input.! Please Try Again."<<endl;
		}
	}
	return 0;
}
